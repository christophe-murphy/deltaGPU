#include "hip/hip_runtime.h"
#include "DeltaGPU.cuh"
#include "Camera.cuh"

__global__ void ray_trace_kernel(Model model, Camera camera, float* frame_buffer) {
    int ray_id = blockIdx.x * blockDim.x + threadIdx.x;
    if (ray_id >= camera.px_width * camera.px_height) return;

    int h_pixel = ray_id % camera.px_width;
    int v_pixel = ray_id / camera.px_width;

    frame_buffer[ray_id] = camera.ray_trace(model, h_pixel, v_pixel);
}

__host__ DeltaGPU::DeltaGPU(const char* model_file_name) {
    num_rays = 0;
    Facet* facets, facets_dev;
    int n_facets = construct_model(facets, model_file_name);
    hipMalloc(&facets_dev, n_facets * sizeof(Facet));
    hipMemcpy(facets_dev, facets, n_facets * sizeof(Facet), hipMemcpyHostToDevice);
    model = Model(facets_dev, n_facets);
}

__host__ DeltaGPU::~DeltaGPU() {
    hipFree(model.facets);
}

__host__ int DeltaGPU::construct_model(Facet* facets, const char* model_file_name) const {
    //Stuff for reading stl file
}

__host__ void DeltaGPU::allocate_framebuffer(float* frame_buffer, int num_rays_in) {
    if (num_rays_in != num_rays) {
        if (num_rays != 0) {
	    hipFree(frame_buffer);
	}
	num_rays = num_rays_in;
	hipMalloc(&frame_buffer, num_rays * sizeof(float));
    }
}

__host__ void DeltaGPU::ray_trace(const int px_width, const int px_height, float* position,
		                  float* target, float* up, float* frame_buffer) {

    float3 posn(position[0], position[1], position[2]);
    float3 targ(target[0], target[1], target[2]);
    float3 u(up[0], up[1], up[2]);
    Camera camera(px_width, px_height, posn, targ, u);
    allocate_framebuffer(frame_buffer_dev, px_width * px_height);
    ray_trace_kernel<<<blocks_per_grid, threads_per_block>>>(model, camera, frame_buffer_dev);
    hipMemcpy(frame_buffer_dev, frame_buffer, num_rays * sizeof(float), hipMemcpyDeviceToHost);
}
